#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void vectAdd( int* a, int* b, int* c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

}

int main()
{
    int a[] = { 1, 2, 3 };
    int b[] = { 4, 5, 6 };
    int c[sizeof(a) / sizeof(int)] = {0};

    int* cA = 0;
    int* cB = 0;
    int* cC = 0;

    hipMalloc(&cA, sizeof(a));
    hipMalloc(&cB, sizeof(b));
    hipMalloc(&cC, sizeof(c));

    hipMemcpy(cA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cB, b, sizeof(b), hipMemcpyHostToDevice);
    hipMemcpy(cC, c, sizeof(c), hipMemcpyHostToDevice);

    vectAdd<<< 1, sizeof(a) / sizeof(int) >>> (cA, cB, cC);

    hipMemcpy(c, cC, sizeof(c), hipMemcpyDeviceToHost);

    for(int i = 0; i < (sizeof(a) / sizeof(int)); i++)
    {
        printf("%d,",c[i]);
    }

    return 0;
}

